#include "hip/hip_runtime.h"
#include "filters_gpu.h"


////// GAUSSIAN BLUR //////

__global__ void gaussianBlurKernel(const uchar* input, uchar* output, int rows, int cols, int kWidth, int kHeight) {
	// PLACEHOLDER
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        output[idx] = input[idx]; 
    }
}

hipError_t launchGaussianBlur(const uchar* d_input, uchar* d_output, int rows, int cols, cv::Size kernelSize, double sigma, dim3 grid, dim3 block)
{
	gaussianBlurKernel <<<grid, block >>>(d_input, d_output, rows, cols, kernelSize.width, kernelSize.height);
	return hipSuccess;
}


////// EROSION //////

__global__ void erosionKernel(const uchar* input, uchar* output, int rows, int cols, const uchar* mask, int kWidth, int kHeight) {
	// PLACEHOLDER
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < cols && y < rows) {
		int idx = y * cols + x;
		output[idx] = input[idx];
	}
}

hipError_t launchErosion(const uchar* d_input, uchar* d_output, int rows, int cols, const uchar* d_mask, cv::Size kernelSize, dim3 grid, dim3 block)
{
	erosionKernel<<<grid, block >>>(d_input, d_output, rows, cols, d_mask, kernelSize.width, kernelSize.height);
	return hipSuccess;
}


////// DILATION //////

__global__ void dilationKernel(const uchar* input, uchar* output, int rows, int cols, const uchar* mask, int kWidth, int kHeight) {
	// PLACEHOLDER
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < cols && y < rows) {
		int idx = y * cols + x;
		output[idx] = input[idx];
	}
}

hipError_t launchDilation(const uchar* d_input, uchar* d_output, int rows, int cols, const uchar* d_mask, cv::Size kernelSize, dim3 grid, dim3 block)
{
	dilationKernel <<<grid, block >>>(d_input, d_output, rows, cols, d_mask, kernelSize.width, kernelSize.height);
	return hipSuccess;
}

////// OPENING //////

__global__ void openingKernel(const uchar* input, uchar* output, int rows, int cols, const uchar* mask, int kWidth, int kHeight) {
	// PLACEHOLDER
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < cols && y < rows) {
		int idx = y * cols + x;
		output[idx] = input[idx];
	}
}

hipError_t launchOpening(const uchar* d_input, uchar* d_output, int rows, int cols, const uchar* d_mask, cv::Size kernelSize, dim3 grid, dim3 block)
{
	openingKernel <<<grid, block >>>(d_input, d_output, rows, cols, d_mask, kernelSize.width, kernelSize.height);
	return hipSuccess;
}

////// CLOSING //////

__global__ void closingKernel(const uchar* input, uchar* output, int rows, int cols, const uchar* mask, int kWidth, int kHeight) {
	// PLACEHOLDER
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < cols && y < rows) {
		int idx = y * cols + x;
		output[idx] = input[idx];
	}
}

hipError_t launchClosing(const uchar* d_input, uchar* d_output, int rows, int cols, const uchar* d_mask, cv::Size kernelSize, dim3 grid, dim3 block)
{
	closingKernel <<<grid, block >>>(d_input, d_output, rows, cols, d_mask, kernelSize.width, kernelSize.height);
	return hipSuccess;
}