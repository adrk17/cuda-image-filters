#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>

// Macro for checking CUDA errors
#define CUDA_CHECK(call)                                                        \
    do {                                                                        \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__        \
                      << " - " << hipGetErrorString(err) << std::endl;         \
            std::exit(EXIT_FAILURE);                                            \
        }                                                                       \
    } while (0)

#define TILE_WIDTH 16


__global__ void kernel(int *c, const int *a, const int *b)
{

}

int main() {
    cv::Mat img(100, 100, CV_8UC1, cv::Scalar(128));
    cv::imshow("Test", img);
    cv::waitKey(0);
    return 0;
}

