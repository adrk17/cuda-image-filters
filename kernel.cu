#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>

#include "image_loader.h"
#include "filter_types.h"
#include "filter_interface.h"

void testEveryKernel(FilterParams& params, cv::Mat image, bool showImg = true);
void showResizedIfNeeded(const std::string& winName, const cv::Mat& img, int maxWidth = 1000, int maxHeight = 1200);

int main() {
    cv::utils::logging::setLogLevel(cv::utils::logging::LOG_LEVEL_WARNING);

    std::string path = "./galaxy.jpeg";
    cv::Mat input = ImageLoader::loadImage(path);

    FilterParams params;
	params.kernelWidth = 9;
	params.sigma = 5.0f;
	params.morphShape = cv::MORPH_CROSS;
	params.morphKernelSize = cv::Size(8, 5); 

   /* FilterType type = FilterType::CLOSING;

	cv::imshow("Input", input);
	cv::Mat outputCpu = applyFilterCpu(input, type, params);
	cv::imshow("Output CPU", outputCpu);
	cv::Mat outputGpu = applyFilterGpu(input, type, params);
	cv::imshow("Output GPU", outputGpu);
	cv::imwrite("output_cpu.png", outputCpu);
	cv::imwrite("output_gpu.png", outputGpu);
	checkDifferance(outputCpu, outputGpu);
	
	cv::waitKey(0);*/

	testEveryKernel(params, input, true);
    return 0;
}

void checkDifferance(cv::Mat& img1, cv::Mat& img2, bool showImg) {
	if (img1.size() != img2.size()) {
		std::cerr << "Images are not the same size!" << std::endl;
	}

	cv::Mat diff;
	cv::absdiff(img1, img2, diff);
	cv::Mat diffDisplay;
	diff.convertTo(diffDisplay, CV_8U, 10); // alpha - scale the difference for better visibility
	if (showImg)
		showResizedIfNeeded("Difference", diffDisplay);

	double minVal, maxVal;
	cv::minMaxLoc(diff, &minVal, &maxVal);
	cv::Scalar meanDiff = cv::mean(diff);

	cv::Mat sorted;
	cv::sort(diff.reshape(1, 1), sorted, cv::SORT_EVERY_ROW + cv::SORT_ASCENDING);
	uchar medianVal = sorted.at<uchar>(sorted.cols / 2);

	std::cout << "Min difference: " << minVal << std::endl;
	std::cout << "Max difference: " << maxVal << std::endl;
	std::cout << "Average difference: " << meanDiff[0] << std::endl;
	std::cout << "Median difference: " << static_cast<int>(medianVal) << std::endl;



	int totalPixels = diff.total();
	int nonZeroPixels = cv::countNonZero(diff);
	std::cout << "Non-zero different pixels: " << nonZeroPixels << " / " << totalPixels << std::endl;

	int histSize = 256;
	float range[] = { 0, 256 };
	const float* histRange = { range };
	cv::Mat hist;
	cv::calcHist(&diff, 1, 0, cv::Mat(), hist, 1, &histSize, &histRange);

	std::cout << "\n--- Differance Histogram ---\n";
	for (int i = 0; i < 256; ++i) {
		int count = cvRound(hist.at<float>(i));
		if (count > 0) {
			std::cout << "Differance " << i << ": " << count << std::endl;
		}
	}
}

void testEveryKernel(FilterParams& params, cv::Mat image, bool showImg) {
	std::vector<std::tuple<FilterType, std::string>> filterTypes = {
		{FilterType::GAUSSIAN_BLUR, "Gaussian Blur"},
		{FilterType::EROSION, "Erosion"},
		{FilterType::DILATION, "Dilation"},
		{FilterType::OPENING, "Opening"},
		{FilterType::CLOSING, "Closing"}
	};


	for (auto filterTuple : filterTypes)
	{
		FilterType type;
		std::string filterName;
		std::tie(type, filterName) = filterTuple;
		std::cout << "\n================ " << filterName << " ================\n\n";
		cv::Mat outputCpu = applyFilterCpu(image, type, params);
		cv::Mat outputGpu = applyFilterGpu(image, type, params);
		
		checkDifferance(outputCpu, outputGpu, showImg);
		if (showImg)
		{
			showResizedIfNeeded("Output CPU - " + filterName, outputCpu);
			showResizedIfNeeded("Output GPU - " + filterName, outputGpu);
			cv::imwrite("output_cpu_" + filterName + ".png", outputCpu);
			cv::imwrite("output_gpu_" + filterName + ".png", outputGpu);
			cv::waitKey(0);
		}

		std::cout << "\n================ " << filterName << " ================\n\n\n\n";
	}
}

void showResizedIfNeeded(const std::string& winName, const cv::Mat& img, int maxWidth, int maxHeight) {
	cv::Mat display = img;

	if (img.cols > maxWidth || img.rows > maxHeight) {
		double scaleW = static_cast<double>(maxWidth) / img.cols;
		double scaleH = static_cast<double>(maxHeight) / img.rows;
		double scale = std::min(scaleW, scaleH);

		cv::resize(img, display, cv::Size(), scale, scale);
	}

	cv::imshow(winName, display);
}