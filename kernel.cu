#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>

#include "image_loader.h"
#include "filter_types.h"
#include "filter_interface.h"

/**
 * @brief Tests all supported filters (Gaussian blur, erosion, dilation, opening, closing) on the provided image.
 *
 * For each filter type, applies the filter using OpenCV CPU, custom CUDA, and OpenCV CUDA implementations.
 * Compares the results and, if verbose is enabled, displays and saves the output images.
 *
 * @param params   Filter parameters to use for all filters.
 * @param image    Input image to filter.
 * @param verbose  Verbosity level (0 = silent, 1 = log info, 2 = log info and show images).
 */
void testEveryKernel(FilterParams& params, cv::Mat image, int verbose = 1);
/**
 * @brief Displays an image in a window, resizing it if it exceeds the specified maximum dimensions.
 *
 * If the image is larger than maxWidth or maxHeight, it is scaled down to fit within those bounds.
 * The image is then shown in a window with the given name.
 *
 * @param winName   Name of the display window.
 * @param img       Image to display.
 * @param maxWidth  Maximum allowed width for display (default 1000).
 * @param maxHeight Maximum allowed height for display (default 1200).
 */
void showResizedIfNeeded(const std::string& winName, const cv::Mat& img, int maxWidth = 1000, int maxHeight = 1200);
/**
 * @brief Benchmarks all supported filters by running each for a specified number of iterations.
 *
 * For each filter type, applies the filter using OpenCV CPU, custom CUDA, and OpenCV CUDA implementations.
 * Measures and prints the average execution time for each method. Optionally displays and saves output images.
 *
 * @param params     Filter parameters to use for all filters.
 * @param image      Input image to filter.
 * @param iterations Number of times to run each filter for timing.
 * @param verbose    Verbosity level (0 = silent, 1 = log info, 2 = log info and show images).
 */
void benchmarkEveryKernel(FilterParams& params, cv::Mat image, int iterations, int verbose = 0);
/**
 * @brief Compares two images and reports their differences.
 *
 * Computes absolute difference, min/max/mean/median difference, and counts pixels with significant differences.
 * Separates border and inner differences, and optionally displays the difference image and prints a histogram.
 *
 * @param img1         First image to compare.
 * @param img2         Second image to compare.
 * @param verbose      Verbosity level (0 = summary, 1 = detailed, 2 = show difference image).
 * @param borderSize   Size of the border region to consider for border differences.
 * @param diffThreshold Pixel difference threshold to count as significant.
 */
void checkDifferance(const cv::Mat& img1, const cv::Mat& img2, int verbose = 1, int borderSize = 10, int diffThreshold = 0);


/**
 * @brief Main entry point of the program.
 *
 * Loads an input image, sets up filter parameters, and benchmarks all supported filters
 * (Gaussian blur, erosion, dilation, opening, closing) using CPU, custom CUDA, and OpenCV CUDA implementations.
 * Prints average execution times for each method.
 *
 * @return 0 on successful execution.
 */
int main() {
	cv::utils::logging::setLogLevel(cv::utils::logging::LOG_LEVEL_WARNING);
	//std::cout << "CUDA devices: " << cv::cuda::getCudaEnabledDeviceCount() << std::endl;

	std::string path = "./galaxy.jpeg";
	cv::Mat input = ImageLoader::loadImage(path);

	FilterParams params;
	params.kernelWidth = 9;
	params.sigma = 5.0f;
	params.morphShape = cv::MORPH_CROSS;
	params.morphKernelSize = cv::Size(8, 5);

	benchmarkEveryKernel(params, input, 5, 0);

	return 0;
}

void testEveryKernel(FilterParams& params, cv::Mat image, int verbose) {
	std::vector<std::tuple<FilterType, std::string>> filterTypes = {
		{FilterType::GAUSSIAN_BLUR, "Gaussian Blur"},
		{FilterType::EROSION, "Erosion"},
		{FilterType::DILATION, "Dilation"},
		{FilterType::OPENING, "Opening"},
		{FilterType::CLOSING, "Closing"}
	};


	for (auto filterTuple : filterTypes)
	{
		FilterType type;
		std::string filterName;
		std::tie(type, filterName) = filterTuple;
		std::cout << "\n================ " << filterName << " ================\n\n";
		cv::Mat outputCpu = applyFilterOpenCvCpu(image, type, params, verbose > 0);
		cv::Mat outputGpu = applyFilterGpu(image, type, params, verbose > 0);
		cv::Mat outputGpuOpenCv = applyFilterOpenCvGpu(image, type, params, verbose > 0);

		checkDifferance(outputCpu, outputGpu, verbose);
		if (verbose > 1)
		{
			showResizedIfNeeded("Output CPU - " + filterName, outputCpu);
			showResizedIfNeeded("Output GPU - " + filterName, outputGpu);
			cv::imwrite("output_cpu_" + filterName + ".png", outputCpu);
			cv::imwrite("output_gpu_" + filterName + ".png", outputGpu);
			cv::waitKey(0);
		}

		std::cout << "\n================ " << filterName << " ================\n\n\n\n";
	}
}

void showResizedIfNeeded(const std::string& winName, const cv::Mat& img, int maxWidth, int maxHeight) {
	cv::Mat display = img;

	if (img.cols > maxWidth || img.rows > maxHeight) {
		double scaleW = static_cast<double>(maxWidth) / img.cols;
		double scaleH = static_cast<double>(maxHeight) / img.rows;
		double scale = std::min(scaleW, scaleH);

		cv::resize(img, display, cv::Size(), scale, scale);
	}

	cv::imshow(winName, display);
}

void benchmarkEveryKernel(FilterParams& params, cv::Mat image, int iterations, int verbose) // 0 - no logs, 1 - logs in filter interface, 2-  logs in filter interface + show images
{
	std::vector<std::tuple<FilterType, std::string>> filterTypes = {
		{FilterType::GAUSSIAN_BLUR, "Gaussian Blur"},
		{FilterType::EROSION, "Erosion"},
		{FilterType::DILATION, "Dilation"},
		{FilterType::OPENING, "Opening"},
		{FilterType::CLOSING, "Closing"}
	};
	for (auto filterTuple : filterTypes)
	{
		FilterType type;
		std::string filterName;
		std::tie(type, filterName) = filterTuple;
		std::cout << "\n================ " << filterName << " ================\n\n";

		std::cout << "Average time for " << iterations << " iterations:\n";
		float avgMsCpu = 0.0f;
		cv::Mat outputCpu = applyFilterOpenCvCpuIterations(image, type, params, iterations, verbose > 0, &avgMsCpu);
		std::cout << "[CPU-OpenCV]\t:\t" << avgMsCpu << " ms\n";
		float avgMsGpu = 0.0f;
		cv::Mat outputGpu = applyFilterGpuIterations(image, type, params, iterations, verbose > 0, &avgMsGpu);
		std::cout << "[GPU-Custom]\t:\t" << avgMsGpu << " ms\n";
		float avgMsGpuOpenCv = 0.0f;
		cv::Mat outputGpuOpenCv = applyFilterOpenCvGpuIterations(image, type, params, iterations, verbose > 0, &avgMsGpuOpenCv);
		std::cout << "[GPU-OpenCV]\t:\t " << avgMsGpuOpenCv << " ms\n\n\n";

		checkDifferance(outputCpu, outputGpu, verbose);

		/*std::cout << "Checking OpenCV GPU results...\n";
		checkDifferance(outputCpu, outputGpuOpenCv, 0);*/
		if (verbose > 1)
		{
			showResizedIfNeeded("Output CPU - " + filterName, outputCpu);
			showResizedIfNeeded("Output GPU - " + filterName, outputGpu);
			cv::imwrite("output_cpu_" + filterName + ".png", outputCpu);
			cv::imwrite("output_gpu_" + filterName + ".png", outputGpu);
			cv::waitKey(0);
		}
		std::cout << "\n================ " << filterName << " ================\n\n\n\n";
	}
}

void checkDifferance(const cv::Mat& img1, const cv::Mat& img2, int verbose, int borderSize, int diffThreshold) {
	CV_Assert(img1.size() == img2.size() && img1.type() == img2.type());

	cv::Mat diff;
	cv::absdiff(img1, img2, diff);

	if (verbose > 1) {
		cv::Mat diffDisplay;
		diff.convertTo(diffDisplay, CV_8U, 10); // scale difference
		showResizedIfNeeded("Difference", diffDisplay);
	}

	double minVal, maxVal;
	cv::minMaxLoc(diff, &minVal, &maxVal);
	cv::Scalar meanDiff = cv::mean(diff);

	cv::Mat sorted;
	cv::sort(diff.reshape(1, 1), sorted, cv::SORT_EVERY_ROW + cv::SORT_ASCENDING);
	uchar medianVal = sorted.at<uchar>(sorted.cols / 2);

	if (verbose > 0) {
		std::cout << "Min difference: " << minVal << "\n";
		std::cout << "Max difference: " << maxVal << "\n";
		std::cout << "Average difference: " << meanDiff[0] << "\n";
		std::cout << "Median difference: " << static_cast<int>(medianVal) << "\n";
	}

	int totalPixels = diff.total();
	int nonZeroPixels = cv::countNonZero(diff > diffThreshold);

	// Count inner vs border differences
	int borderDiffs = 0, innerDiffs = 0;
	for (int y = 0; y < diff.rows; ++y) {
		for (int x = 0; x < diff.cols; ++x) {
			if (diff.at<uchar>(y, x) > diffThreshold) {
				bool isBorder = (x < borderSize || x >= diff.cols - borderSize ||
					y < borderSize || y >= diff.rows - borderSize);
				if (isBorder)
					borderDiffs++;
				else
					innerDiffs++;
			}
		}
	}

	if (verbose > -1) {
		std::cout << "Total different pixels (> " << diffThreshold << "): " << nonZeroPixels << " / " << totalPixels << "\n";
		std::cout << " - Border differences: " << borderDiffs << "\n";
		std::cout << " - Inner differences: " << innerDiffs << "\n";
	}

	if (verbose > 0) {
		int histSize = 256;
		float range[] = { 0, 256 };
		const float* histRange = { range };
		cv::Mat hist;
		cv::calcHist(&diff, 1, 0, cv::Mat(), hist, 1, &histSize, &histRange);

		std::cout << "\n--- Difference Histogram ---\n";
		for (int i = 0; i < 256; ++i) {
			int count = cvRound(hist.at<float>(i));
			if (count > 0) {
				std::cout << "Difference " << i << ": " << count << "\n";
			}
		}
	}
}